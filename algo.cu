#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

// Expose CUDA functions to the MPI code
extern "C" {
    bool HL_kernelLaunch(int rank, unsigned char** d_data, unsigned char** d_resultData,
                         unsigned char** preData, unsigned char** postData,
                         size_t worldWidth, size_t worldHeight, ushort threadsCount);

    void cudaMallocWrapper(void** ptr, size_t size);
    void cudaFreeWrapper(void* ptr);
}

/**
 * Use CUDA malloc to allocate memory on the device.
 * @param ptr The pointer to the memory to allocate.
 * @param size The size of the memory to allocate.
 */
void cudaMallocWrapper(void** ptr, size_t size) {
    hipMallocManaged(ptr, size);
}

/**
 * Use CUDA free to deallocate memory on the device.
 * @param ptr The pointer to the memory to deallocate.
 */
void cudaFreeWrapper(void* ptr) {
    hipFree(ptr);
}


/**
 * Swap the pointers of two arrays.
 * @param pA The first array.
 * @param pB The second array.
 */
static inline void HL_swap( unsigned char **pA, unsigned char **pB) {
  unsigned char *temp = *pA;
  *pA = *pB;
  *pB = temp;
}


/**
 * Count the number of alive cells in the neighborhood of a cell.
 * @param data The current state of the world.
 * @param preData The previous ghost row.
 * @param postData The next ghost row.
 * @param x0 The x coordinate of the first column of cells.
 * @param x1 The x coordinate of the second column of cells.
 * @param x2 The x coordinate of the third column of cells.
 * @param y0 The y coordinate of the first row of cells.
 * @param y1 The y coordinate of the second row of cells.
 * @param y2 The y coordinate of the third row of cells.
 * @return The number of alive cells in the neighborhood.
 */
__device__ unsigned int HL_countAliveCells(const unsigned char* data, unsigned char* preData, unsigned char* postData,
					   size_t x0,
					   size_t x1,
					   size_t x2,
					   size_t y0,
					   size_t y1,
					   size_t y2) {

    unsigned int topSum = 0;
    unsigned int botSum = 0;

    // If y0 does not wrap around: set the top sum normally, otherwise: set the top sum based on the previous ghost row
    if (y0 < y1) topSum += data[x0 + y0] + data[x1 + y0] + data[x2 + y0];
    else topSum += preData[x0] + preData[x1] + preData[x2];

    // If y2 does not wrap around: set the bottom sum normally, otherwise: set the bottom sum based on the next ghost row
    if (y2 > y1) botSum += data[x0 + y2] + data[x1 + y2] + data[x2 + y2];
    else botSum += postData[x0] + postData[x1] + postData[x2];

    // Return the final sum of the neighborhood
    return topSum + data[x0 + y1] + data[x2 + y1] + botSum;
}

/**
 * The kernel for running HighLife in parallel on a GPU.
 * @param d_data The current state of the world.
 * @param d_resultData The next state of the world.
 * @param worldWidth The width of the world.
 * @param worldHeight The height of the world.
 */
__global__ void HL_kernel(int rank, const unsigned char* d_data, unsigned char* d_resultData,
                          unsigned char* preData, unsigned char* postData, size_t worldWidth, size_t worldHeight, int cell_dim) {
    int device;
    hipGetDevice( &device );

    // Calculate the index of the cell in the world, striding by the total number of threads in the grid

    // **Use size_t type to avoid overflows when repeatedly adding**
    // Individual variables (blockIdx.x, etc.) likely do not need to be cast to size_t, but memory is not very constrained, so it can't hurt
    // Cast anyways for compatibility
    for(size_t index = (size_t) blockIdx.x * (size_t) blockDim.x + (size_t) threadIdx.x;
            index < worldWidth*worldHeight;
            index += (size_t) blockDim.x * (size_t) gridDim.x) {


        size_t trueWidth = worldWidth * cell_dim;
        // Calculate the x and y coordinates of the cell based on the global index
        size_t x = index % trueWidth;
        size_t y = index / trueWidth;

        // Calculate the surrounding cells exactly like the serial program
        size_t y0 = ((y + worldHeight - 1) % worldHeight) * trueWidth;
        size_t y1 = y * trueWidth;
        size_t y2 = ((y + 1) % worldHeight) * trueWidth;

        size_t x0 = (x + trueWidth - cell_dim) % trueWidth;
        size_t x2 = (x + cell_dim) % trueWidth;

        // Count the number of alive cells in the neighborhood
        unsigned int cityChance = rateSpot(d_data, preData, postData, cell_dim, x0, x, x2, y0, y1, y2);
        // rule B36/S23
        // Set the next state of the cell based on the number of alive cells in the neighborhood
        d_resultData[x + y1] = (aliveCells == 3) || (aliveCells == 6 && !d_data[x + y1])
            || (aliveCells == 2 && d_data[x + y1]) ? 1 : 0;
    }
}

/**
 * Launch the kernel once for each iteration, synchronizing after each launch.
 * @param d_data The current state of the world.
 * @param d_resultData The next state of the world.
 * @param worldWidth The width of the world.
 * @param worldHeight The height of the world.
 * @param iterationsCount The number of iterations to run the algorithm.
 * @param threadsCount The number of threads to allocate to each block in the kernel.
 * @return True if the kernel was launched successfully.
 */
bool HL_kernelLaunch(int rank, unsigned char** d_data, unsigned char** d_resultData,
                     unsigned char** preData, unsigned char** postData,
                     size_t worldWidth, size_t worldHeight, int cell_dim, ushort threadsCount) {

    // Get the number of cuda devices and set the current device to the rank modulo the number of devices
    hipError_t hipError_t;
    int cudaDeviceCount;
    if( (hipError_t = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess ) {
        printf(" Unable to determine cuda device count, error is %d, count is %d\n", hipError_t, cudaDeviceCount );
        exit(-1);
    }
    if( (hipError_t = hipSetDevice( rank % cudaDeviceCount )) != hipSuccess ) {
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n", rank, (rank % cudaDeviceCount), hipError_t);
        exit(-1);
    }

    // Determine how many blocks should be allocated to the kernel with a maximum of 65535
    size_t blockCount = (worldHeight * worldWidth * cell_dim) / threadsCount + 1;
    blockCount = blockCount > 65535 ? 65535 : blockCount;
    // Determine the number of threads to allocate to each block in the kernel
    size_t kernelThreads = threadsCount;
    // Launch the kernel with the determined block count and thread count
    HL_kernel<<<blockCount, kernelThreads>>>(rank, *d_data, *d_resultData, *preData, *postData, worldWidth, worldHeight, cell_dim);
    // Synchronize the device after each launch
    hipDeviceSynchronize();

    // Swap the pointers of the current state and the next state of the world
    HL_swap(d_data, d_resultData);

    return true;
}


int slopeThresh = 20;
float basePercent = 50;
int *slopePenalty;

int *tempMin;
int *tempMax;
int *tempPenalty;

int *rainMax;
int *rainMin;
int *rainPenalty;

float waterBuff;

/* short slope, 
short temp, 
short rain, 
short elevation, 
short water, 
short resources, 
short biome */


//preData is the ghost row before
//postData is the ghost row after
int rateSpot(char* d_data, char* preData, char* postData, int cell_dim, size_t x0, size_t x, size_t x2, size_t y0, size_t y1, size_t y2);{
    int cityChance = basePercent;

    short slope = d_data[x];
    short temp = d_data[x+1];
    short rainfall = d_data[x+2];
    short elevation = d_data[x+3];
    short water = d_data[x+4];
    short[] neighborWater = [d_data[y0+x0+4], d_data[y0+x+4], d_data[y0+x2+4], d_data[y1+x0+4], d_data[y1+x2+4], d_data[y2+x0+4], d_data[y2+x+4], d_data[y2+x2+4],]
    short resource = d_data[x+5];
    short biome = d_data[x+6];

    //being underwater is bad
    if(water != 0){
        return 0;
    }

    //count the number of neighbors that are underwater. Coastline, good, peninsula fine, island questionable
    short neighborsUnderwater = 0;
    for(int i = 0; i<neighborWater.size(); i++){
        if(neighborWater[i] != 0){
            neighborsUnderwater += 1;
        }
    }

    //add buffs accordingly
    if (neighborsUnderwater > 5){
        cityChance += *peninsulaBuff;
    }
    elif(neighborsUnderwater > 2){
        cityChance += *coastBuff;
    }
    elif(neighborsUnderwater > 0){
        cityChance += *waterBuff;
    }

    //add up the elevation differences between this cell and its neighbors, if greater than slopeThresh, sub
    float slopeSum = 0;
    for(int i = 0; i<slope.size(); i++){
        slopeSum += abs(slope[i] - elevation);
    }
    if(slopeSum > slopeThresh){
        cityChance -= (slopeSum * (*slopePenalty));
    }

    //very hot or very cold is bad
    if(temp > *tempMax || temp < *tempMin){
        cityChance -= *tempPenalty;
    }    

    //rainforest and desert are bad
    if(rain > *rainMax || rain < *rainMin){
        cityChance -= *rainPenalty;
    }

    //resources good
    for(int r = 0; r<resources.size(); r++){
        cityChance += resourceMap(resources[r]);
    }


    return cityChance; 

}