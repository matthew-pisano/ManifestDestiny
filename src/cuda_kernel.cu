#include "hip/hip_runtime.h"
//
// Created by matthew on 4/9/24.
//


#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>

#include "../include/data_rep.h"


/**
 * Use CUDA malloc managed to allocate memory on the device.
 * @param ptr The pointer to the memory to allocate.
 * @param size The size of the memory to allocate.
 */
extern "C" void __cudaMalloc(void** ptr, size_t size) {
    hipMallocManaged(ptr, size);
}

/**
 * Use CUDA memcpy to copy memory between the host and device.
 * @param dst The destination pointer to copy to.
 * @param src The source pointer to copy from.
 * @param size The size of the memory to copy.
 * @param kind The direction of the copy.
 */
extern "C" void __cudaMemcpy(void* dst, void* src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyDefault);
}

/**
 * Use CUDA free to deallocate memory on the device.
 * @param ptr The pointer to the memory to deallocate.
 */
extern "C" void __cudaFree(void* ptr) {
    hipFree(ptr);
}


struct Neighborhood {
    unsigned short count;
    unsigned short max;
    unsigned short avg;
    unsigned short min;
};


/**
 * Count the values of the neighbors of a cell, within a given radius
 * @param target_index The index of the cell to count the neighbors of
 * @param radius The radius of the neighborhood to count
 * @param data_dims The dimensions of the data
 * @param ghost_cols The ghost columns for the data
 * @param data The buffer containing the data
 * @return The count of the values of the neighbors
 */
__device__ Neighborhood count_neighbor_values(int target_index, int radius, struct DataDims data_dims, struct GhostCols ghost_cols, unsigned short *data) {
    int cells_seen = 0;
    int count = 0;
    unsigned short max_value = 0;
    unsigned short min_value = 65535;
    int col_len = data_dims.col_dim * data_dims.cell_dim;
    int world_len = col_len * data_dims.row_dim;
    // The offset of the given target cell from a cell boundary is maintained in the calculations.
    // If the target is offset by 7 from a cell boundary, all counts will be offset by 7 within the neighboring cells.
    int pos_in_col = target_index % col_len;

    // Count for all cells within the radius of the target cell
    for (int x_offset=-radius*col_len; x_offset<=radius*col_len; x_offset+=col_len) {
        for (int y_offset=-radius*data_dims.cell_dim; y_offset<=radius*data_dims.cell_dim; y_offset+=data_dims.cell_dim) {

            int neighbor_index = target_index + x_offset + y_offset;
            bool is_y_bounded = pos_in_col + y_offset >= 0 && pos_in_col + y_offset < col_len;
            unsigned short cell_value = 0;
            if (neighbor_index >= 0 && neighbor_index < world_len && neighbor_index != target_index && is_y_bounded) {
                cell_value = data[neighbor_index];
                cells_seen++;
            }
            else if (neighbor_index < 0 && is_y_bounded && ghost_cols.west != NULL) {
                cell_value = ghost_cols.west[pos_in_col + y_offset];
                cells_seen++;
            }
            else if (neighbor_index >= world_len && is_y_bounded && ghost_cols.east != NULL) {
                cell_value = ghost_cols.east[pos_in_col + y_offset];
                cells_seen++;
            }

            count += cell_value;
            if (cell_value > max_value) max_value = cell_value;
            if (cell_value < min_value) min_value = cell_value;
        }
    }
    // Ensure the count is within the bounds of an unsigned short
    count = count > 65535 ? 65535 : count;

    unsigned short avg = (unsigned short) (count / cells_seen);

    struct Neighborhood neighborhood = {(unsigned short)count, max_value, avg, min_value};
    return neighborhood;
}


__device__ int generate_jitter(int target_cell, int iteration, int max_jitter) {
    int seed = target_cell * iteration;

    int a = 16807;
    int m = 2147483647;
    seed = (a * seed) % m;
    return seed % max_jitter;
}

/**
 * Get the new population of a cell based on the values of its neighbors
 * @param target_cell The base (zero) index of the cell to update
 * @param data_dims The dimensions of the data
 * @param ghost_cols The ghost columns for the data
 * @param data The buffer containing the data
 * @return The new population of the cell
 */
__device__ unsigned short calc_cell_population(int target_cell, int iteration, struct DataDims data_dims, struct GhostCols ghost_cols, unsigned short *data) {

    unsigned short elev = data[target_cell];
    unsigned short grad = data[target_cell+1];
    unsigned short water = data[target_cell+2];
    unsigned short temp = data[target_cell+3];
    unsigned short precip = data[target_cell+4];
    unsigned short resources = data[target_cell+5];
    unsigned short biome = data[target_cell+6];
    unsigned short pop = data[target_cell+7];

    const unsigned short RES_SCALE = data_dims.col_dim / 623;
    const unsigned short RES_SCALE_SQ = RES_SCALE * RES_SCALE;
    const unsigned short MIN_POP = (2 * 9) / RES_SCALE_SQ;

    const int MAX_JITTER = 10000;

    //set up the random number generator
    //hiprandStateMRG32k3a_t* rand_state = NULL;
    //"seed", "sequence", "offset", the random state pointer
    //hiprand_init(500,500,500,rand_state);
    int jitter = generate_jitter(target_cell, iteration, MAX_JITTER);// hiprand(rand_state) % MAX_JITTER;
    float jitter_range = (jitter - MAX_JITTER / 2.0) / MAX_JITTER;

    //printf("Cell stats: elev: %d, grad: %d, water: %d, temp: %d, precip: %d, resources: %d, biome: %d, pop: %d\n", elev, grad, water, temp, precip, resources, biome, pop);

    // If the cell is water, too high, too steep, or too dry then return 0
    if (water > 0 || elev > 10000 || grad > 30) return 0;

    // If the cell is too populous then return 0
    if (pop > 40000) return pop;

    // Value of a cell on a scale of 0-100
    short cell_value = jitter_range * 30;
    unsigned short nearby_water = count_neighbor_values(target_cell+2, 2*RES_SCALE, data_dims, ghost_cols, data).max;

    cell_value += nearby_water * 3;
    cell_value += resources;
    cell_value += 15 - (elev / 1500);
    cell_value += temp > 45 && temp < 70 ? 15 : 0;
    cell_value += precip > 30 ? 10 : precip < 10 ? -10 : 0;
    // If the biome is a swamp
    if (biome == 4) cell_value -= 10;
    // If the biome is a desert
    if (biome == 5) cell_value -= 15;

    // Clamp the cell value to 0-200
    if (cell_value < 0) cell_value = 0;
    if (cell_value > 200) cell_value = 200;

    struct Neighborhood nearby_population = count_neighbor_values(target_cell+7, 2*RES_SCALE, data_dims, ghost_cols, data);

    // ~~~ EXPLORATION PHASE ~~~ //

    // If the cell is uninhabited and the cell value is high enough, explore the cell
    float explore_chance = 0.002 + (iteration * iteration) / 8000000.0;

    if (pop == 0 && nearby_population.count > 0)
        return jitter < explore_chance * (cell_value / 100.0 + 3) * MAX_JITTER ? MIN_POP : 0;
    else if (pop == 0) return 0;

    // ~~~ SETTLEMENT PHASE ~~~ //

    // If the cell is explored with no nearby cities
    if (pop == MIN_POP && nearby_population.avg <= MIN_POP / 1.5) {
        // Settle new city
        if (jitter < 8) return 8 * MIN_POP * RES_SCALE_SQ;
    }
    // If the cell is settled with nearby cities, expand the city with this cell
    else if (pop == MIN_POP && jitter < cell_value / 7) {
        if (nearby_population.avg < 150) return MIN_POP + nearby_population.avg / 3 * RES_SCALE_SQ;
        else if (nearby_population.avg < 300) return MIN_POP + nearby_population.avg / 4 * RES_SCALE_SQ;
        else if (nearby_population.avg < 500) return MIN_POP + nearby_population.avg / 5 * RES_SCALE_SQ;
        else return MIN_POP + nearby_population.avg / 8 * RES_SCALE_SQ;
    }
    else if (pop == MIN_POP && jitter > MAX_JITTER - cell_value / 20)
        return 1.5 * MIN_POP * RES_SCALE_SQ;

    // Skip growth if the criteria for an already settled cell are not met
    if (pop == MIN_POP) return MIN_POP;

    cell_value -= grad * 2;
    if (cell_value < 0) cell_value = 0;

    // return cell_value * 500;

    // ~~~ GROWTH PHASE ~~~ //

    // Add a bonus to the cell value based on the average population of the neighbors, scaling with the cell value
    unsigned short neighbor_bonus = (cell_value / 100.0) * nearby_population.avg;
    // Grow the cell by a factor of its population, scaling with the cell value, and add the neighbor bonus
    float neighbor_growth_factor = 0.02;
    if (nearby_population.avg > 1500) neighbor_growth_factor = 0;
    else if (nearby_population.avg > 1000) neighbor_growth_factor = 0;
    else if (nearby_population.avg > 750) neighbor_growth_factor = 0.002;
    else if (nearby_population.avg > 500) neighbor_growth_factor = 0.008;

    if (iteration < 100) neighbor_growth_factor *= 0.05;
    else if (iteration < 200) neighbor_growth_factor *= 0.2;
    else if (iteration < 400) neighbor_growth_factor *= 0.4;
    else if (iteration < 600) neighbor_growth_factor *= 0.6;

    float city_center_bonus = 1;
    if (pop * (1+jitter_range*0.5) > nearby_population.avg) city_center_bonus = 2.1;

    float growth_factor = 0.011 * city_center_bonus;
    if (pop > 3500) growth_factor *= 0.07;
    else if (pop > 2500) growth_factor *= 0.1;
    else if (pop > 1500) growth_factor *= 0.11;
    else if (pop > 1000) growth_factor *= 0.12;
    else if (pop > 750) growth_factor *= 0.14;
    else if (pop > 500) growth_factor *= 0.2;
    else if (pop > 300) growth_factor *= 0.3;

    if (iteration < 100) growth_factor *= 0.05;
    else if (iteration < 200) growth_factor *= 0.2;
    else if (iteration < 400) growth_factor *= 0.4;
    else if (iteration < 600) growth_factor *= 0.6;

    neighbor_growth_factor *= RES_SCALE;
    growth_factor /= RES_SCALE_SQ;

    float cell_bonus = cell_value * 1.3;
    if (cell_bonus > 100) cell_bonus = 100;

    return (1 + (cell_bonus / 100.0 * growth_factor)) * pop + neighbor_growth_factor * neighbor_bonus;
}


__global__ void cuda_kernel(int iteration, struct DataDims data_dims, struct GhostCols ghost_cols, unsigned short *data, unsigned short *result_data) {

    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i<data_dims.row_dim * data_dims.col_dim; i+=blockDim.x * gridDim.x) {
        size_t cell_index = i * data_dims.cell_dim;
        unsigned short new_pop = calc_cell_population(cell_index, iteration, data_dims, ghost_cols, data);
	    result_data[cell_index+7] = new_pop;
    }
}


extern "C" void launch_kernel(int iteration, int rank, int thread_count, struct DataDims data_dims, struct GhostCols ghost_cols,
        unsigned short *data, unsigned short *result_data) {

    // Get the number of cuda devices and set the current device to the rank modulo the number of devices
    hipError_t cuda_error;
    int cuda_device_count;
    if( (cuda_error = hipGetDeviceCount( &cuda_device_count)) != hipSuccess ) {
        printf(" Unable to determine cuda device count, error is %d, count is %d\n", cuda_error, cuda_device_count );
        exit(cuda_error);
    }
    if( (cuda_error = hipSetDevice( rank % cuda_device_count )) != hipSuccess ) {
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n", rank, (rank % cuda_device_count), cuda_error);
        exit(cuda_error);
    }

    // Determine how many blocks should be allocated to the kernel with a maximum of 65535
    size_t block_count = (data_dims.row_dim * data_dims.col_dim) / thread_count + 1;
    block_count = block_count > 65535 ? 65535 : block_count;
    // Launch the kernel with the determined block count and thread count
    cuda_kernel<<<block_count, (size_t) thread_count>>>(iteration, data_dims, ghost_cols, data, result_data);
    // Synchronize the device after each launch
    hipDeviceSynchronize();
}
