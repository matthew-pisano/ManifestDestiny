#include "hip/hip_runtime.h"
//
// Created by matthew on 4/9/24.
//

#include "../include/data_rep.h"

#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>


// Linear Congruential Generator (LCG) parameters
#define LCG_M 2147483647 // Modulus
#define LCG_A 1103515245 // Multiplier
#define LCG_C 12345      // Increment


/**
 * Use CUDA malloc managed to allocate memory on the device.
 * @param ptr The pointer to the memory to allocate.
 * @param size The size of the memory to allocate.
 */
extern "C" void __cudaMalloc(void** ptr, size_t size) {
    hipMallocManaged(ptr, size);
}

/**
 * Use CUDA memcpy to copy memory between the host and device.
 * @param dst The destination pointer to copy to.
 * @param src The source pointer to copy from.
 * @param size The size of the memory to copy.
 */
extern "C" void __cudaMemcpy(void* dst, void* src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyDefault);
}

/**
 * Use CUDA free to deallocate memory on the device.
 * @param ptr The pointer to the memory to deallocate.
 */
extern "C" void __cudaFree(void* ptr) {
    hipFree(ptr);
}


/**
 * Struct to hold the count, max, average, and min values of the neighbors of a cell
 */
struct Neighborhood {
    /// The total sum of the values of the neighbors
    unsigned short count;
    /// The maximum value of the neighbors
    unsigned short max;
    /// The average value of the neighbors
    unsigned short avg;
    /// The minimum value of the neighbors
    unsigned short min;
};


/**
 * Count the values of the neighbors of a cell, within a given radius
 * @param target_index The index of the cell to count the neighbors of
 * @param radius The radius of the neighborhood to count
 * @param data_dims The dimensions of the data
 * @param ghost_cols The ghost columns for the data
 * @param data The buffer containing the data
 * @return A struct containing the count, max, average, and min values of the neighbors
 */
__device__ Neighborhood count_neighbor_values(int target_index, int radius, struct DataDims data_dims, struct GhostCols ghost_cols, unsigned short *data) {
    int cells_seen = 0;
    int count = 0;
    unsigned short max_value = 0;
    unsigned short min_value = 65535;
    int col_len = data_dims.col_dim * data_dims.cell_dim;
    int world_len = col_len * data_dims.row_dim;
    // The offset of the given target cell from a cell boundary is maintained in the calculations.
    // If the target is offset by 7 from a cell boundary, all counts will be offset by 7 within the neighboring cells.
    int pos_in_col = target_index % col_len;

    // Count for all cells within the radius of the target cell
    for (int x_offset=-radius*col_len; x_offset<=radius*col_len; x_offset+=col_len) {
        for (int y_offset=-radius*data_dims.cell_dim; y_offset<=radius*data_dims.cell_dim; y_offset+=data_dims.cell_dim) {

            int neighbor_index = target_index + x_offset + y_offset;
            bool is_y_bounded = pos_in_col + y_offset >= 0 && pos_in_col + y_offset < col_len;
            unsigned short cell_value = 0;
            // If the neighbor is within the bounds of the world, get the value of the cell
            if (neighbor_index >= 0 && neighbor_index < world_len && neighbor_index != target_index && is_y_bounded) {
                cell_value = data[neighbor_index];
                cells_seen++;
            }
            // If the neighbor is outside the right bound of the world, get the value from the West ghost column
            else if (neighbor_index < 0 && is_y_bounded && ghost_cols.west != NULL) {
                cell_value = ghost_cols.west[pos_in_col + y_offset];
                cells_seen++;
            }
            // If the neighbor is outside the left bound of the world, get the value from the East ghost column
            else if (neighbor_index >= world_len && is_y_bounded && ghost_cols.east != NULL) {
                cell_value = ghost_cols.east[pos_in_col + y_offset];
                cells_seen++;
            }

            // Update the count, max, and min values
            count += cell_value;
            if (cell_value > max_value) max_value = cell_value;
            if (cell_value < min_value) min_value = cell_value;
        }
    }
    // Ensure the count is within the bounds of an unsigned short
    count = count > 65535 ? 65535 : count;

    unsigned short avg = (unsigned short) (count / cells_seen);

    struct Neighborhood neighborhood = {(unsigned short)count, max_value, avg, min_value};
    return neighborhood;
}


/**
 * Generate a random jitter value based on a seed and a maximum jitter value
 * @param seed The seed to generate the jitter from
 * @param max_jitter The maximum value of the jitter
 * @return The generated jitter value
 */
__device__ int generate_jitter(unsigned int seed, int max_jitter) {

    // Calculate the next seed value using the LCG parameters
    seed = (LCG_A * seed + LCG_C) % LCG_M;
    // Shift jitter by 100 to better match the distribution of the C rand() function
    return (seed-100) % max_jitter;
}

/**
 * Get the new population of a cell based on the values of its neighbors
 * @param target_cell The base (zero) index of the cell to update
 * @param data_dims The dimensions of the data
 * @param ghost_cols The ghost columns for the data
 * @param data The buffer containing the data
 * @return The new population of the cell
 */
__device__ unsigned short calc_cell_population(int target_cell, int iteration, struct DataDims data_dims, struct GhostCols ghost_cols, unsigned short *data) {

    // Get the attributes of the cell
    unsigned short elev = data[target_cell];
    unsigned short grad = data[target_cell+1];
    unsigned short water = data[target_cell+2];
    unsigned short temp = data[target_cell+3];
    unsigned short precip = data[target_cell+4];
    unsigned short resources = data[target_cell+5];
    unsigned short biome = data[target_cell+6];
    unsigned short pop = data[target_cell+7];

    // The current resolution scale of the world
    const unsigned short RES_SCALE = data_dims.col_dim / 623;
    // The square of the resolution scale, used for modifying growth factors
    const unsigned short RES_SCALE_SQ = RES_SCALE * RES_SCALE;
    // The minimum population of a cell
    const unsigned short MIN_POP = (2 * 9) / RES_SCALE_SQ;
    const int MAX_JITTER = 10000;

    // Generate a jitter value based on the target cell and the iteration
    int jitter = generate_jitter(target_cell * iteration, MAX_JITTER);
    float jitter_range = (jitter - MAX_JITTER / 2.0) / MAX_JITTER;

    // If the cell is water, too high, too steep, or too dry then return 0
    if (water > 0 || elev > 10000 || grad > 30) return 0;

    // If the cell is too populous then return 0
    if (pop > 40000) return pop;

    // Value of a cell on a scale of 0-100
    short cell_value = jitter_range * 30;
    unsigned short nearby_water = count_neighbor_values(target_cell+2, 2*RES_SCALE, data_dims, ghost_cols, data).max;

    // ~~~ CALCULATE CELL VALUE ~~~ //

    cell_value += nearby_water * 3;
    cell_value += resources;
    cell_value += 15 - (elev / 1500);
    cell_value += temp > 45 && temp < 70 ? 15 : 0;
    cell_value += precip > 30 ? 10 : precip < 10 ? -10 : 0;
    // If the biome is a swamp
    if (biome == 4) cell_value -= 10;
    // If the biome is a desert
    if (biome == 5) cell_value -= 15;

    // Clamp the cell value to 0-200
    if (cell_value < 0) cell_value = 0;
    if (cell_value > 200) cell_value = 200;

    struct Neighborhood nearby_population = count_neighbor_values(target_cell+7, 2*RES_SCALE, data_dims, ghost_cols, data);

    // ~~~ EXPLORATION PHASE ~~~ //

    // If the cell is uninhabited and the cell value is high enough, explore the cell
    float explore_chance = 0.002 + (iteration * iteration) / 8000000.0;

    if (pop == 0 && nearby_population.count > 0)
        return jitter < explore_chance * (cell_value / 100.0 + 3) * MAX_JITTER ? MIN_POP : 0;
    else if (pop == 0) return 0;

    // ~~~ SETTLEMENT PHASE ~~~ //

    // If the cell is explored with no nearby cities
    if (pop == MIN_POP && nearby_population.avg <= MIN_POP / 1.5) {
        // Settle new city
        if (jitter < 8) return 8 * MIN_POP * RES_SCALE_SQ;
    }
    // If the cell is settled with nearby cities, expand the city with this cell
    else if (pop == MIN_POP && jitter < cell_value / 7) {
        if (nearby_population.avg < 150) return MIN_POP + nearby_population.avg / 3 * RES_SCALE_SQ;
        else if (nearby_population.avg < 300) return MIN_POP + nearby_population.avg / 4 * RES_SCALE_SQ;
        else if (nearby_population.avg < 500) return MIN_POP + nearby_population.avg / 5 * RES_SCALE_SQ;
        else return MIN_POP + nearby_population.avg / 8 * RES_SCALE_SQ;
    }
    else if (pop == MIN_POP && jitter > MAX_JITTER - cell_value / 20)
        return 1.5 * MIN_POP * RES_SCALE_SQ;

    // Skip growth if the criteria for an already settled cell are not met
    if (pop == MIN_POP) return MIN_POP;

    cell_value -= grad * 2;
    if (cell_value < 0) cell_value = 0;

    // return cell_value * 500;

    // ~~~ GROWTH PHASE ~~~ //

    // Add a bonus to the cell value based on the average population of the neighbors, scaling with the cell value
    unsigned short neighbor_bonus = (cell_value / 100.0) * nearby_population.avg;
    // Grow the cell by a factor of its population, scaling with the cell value, and add the neighbor bonus
    float neighbor_growth_factor = 0.02;
    if (nearby_population.avg > 1500) neighbor_growth_factor = 0;
    else if (nearby_population.avg > 1000) neighbor_growth_factor = 0;
    else if (nearby_population.avg > 750) neighbor_growth_factor = 0.002;
    else if (nearby_population.avg > 500) neighbor_growth_factor = 0.008;

    // Scale the neighbor growth factor based on the iteration
    if (iteration < 100) neighbor_growth_factor *= 0.05;
    else if (iteration < 200) neighbor_growth_factor *= 0.2;
    else if (iteration < 400) neighbor_growth_factor *= 0.4;
    else if (iteration < 600) neighbor_growth_factor *= 0.6;

    // Give a bonus to a cell that is the largest in its neighborhood
    float city_center_bonus = 1;
    if (pop * (1+jitter_range*0.5) > nearby_population.avg) city_center_bonus = 2.1;

    // Scale the growth factor based on the population of the cell
    float growth_factor = 0.011 * city_center_bonus;
    if (pop > 3500) growth_factor *= 0.07;
    else if (pop > 2500) growth_factor *= 0.1;
    else if (pop > 1500) growth_factor *= 0.11;
    else if (pop > 1000) growth_factor *= 0.12;
    else if (pop > 750) growth_factor *= 0.14;
    else if (pop > 500) growth_factor *= 0.2;
    else if (pop > 300) growth_factor *= 0.3;

    // Scale the growth factor based on the iteration
    if (iteration < 100) growth_factor *= 0.05;
    else if (iteration < 200) growth_factor *= 0.2;
    else if (iteration < 400) growth_factor *= 0.4;
    else if (iteration < 600) growth_factor *= 0.6;

    // Adjust the growth factors to account for higher resolutions
    neighbor_growth_factor *= RES_SCALE;
    growth_factor /= RES_SCALE_SQ;

    // Cap the cell value at 77, minimizing the growth of the most valuable cells
    float cell_bonus = cell_value * 1.3;
    if (cell_bonus > 100) cell_bonus = 100;

    // Calculate the new population of the cell
    return (1 + (cell_bonus / 100.0 * growth_factor)) * pop + neighbor_growth_factor * neighbor_bonus;
}


__global__ void cuda_kernel(int iteration, struct DataDims data_dims, struct GhostCols ghost_cols, unsigned short *data, unsigned short *result_data) {
    // Stride the kernel to allow for multiple threads to process the data
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i<data_dims.row_dim * data_dims.col_dim; i+=blockDim.x * gridDim.x) {

        size_t cell_index = i * data_dims.cell_dim;
        unsigned short new_pop = calc_cell_population(cell_index, iteration, data_dims, ghost_cols, data);
        // Update the population of the cell in the result data buffer
	    result_data[cell_index+7] = new_pop;
    }
}


extern "C" void launch_kernel(int iteration, int rank, int thread_count, struct DataDims data_dims, struct GhostCols ghost_cols,
        unsigned short *data, unsigned short *result_data) {

    // Get the number of cuda devices and set the current device to the rank modulo the number of devices
    hipError_t cuda_error;
    int cuda_device_count;
    if( (cuda_error = hipGetDeviceCount( &cuda_device_count)) != hipSuccess ) {
        printf(" Unable to determine cuda device count, error is %d, count is %d\n", cuda_error, cuda_device_count );
        exit(cuda_error);
    }
    if( (cuda_error = hipSetDevice( rank % cuda_device_count )) != hipSuccess ) {
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n", rank, (rank % cuda_device_count), cuda_error);
        exit(cuda_error);
    }

    // Determine how many blocks should be allocated to the kernel with a maximum of 65535
    size_t block_count = (data_dims.row_dim * data_dims.col_dim) / thread_count + 1;
    block_count = block_count > 65535 ? 65535 : block_count;
    // Launch the kernel with the determined block count and thread count
    cuda_kernel<<<block_count, (size_t) thread_count>>>(iteration, data_dims, ghost_cols, data, result_data);
    // Synchronize the device after each launch
    hipDeviceSynchronize();
}
