#include "hip/hip_runtime.h"
//
// Created by matthew on 4/9/24.
//

#include "../include/cuda_kernel.cuh"


#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

#include "../include/populate.h"


void cudaMallocManaged_wrapper(void** ptr, size_t size) {
    hipMallocManaged(ptr, size);
}


void cudaFree_wrapper(void* ptr) {
    hipFree(ptr);
}


__global__ void cuda_kernel(struct DataDims data_dims, struct GhostCols ghost_cols, unsigned short *data, unsigned short *result_data) {
    // int device;
    // hipGetDevice(&device);

    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i<data_dims.row_dim * data_dims.col_dim; i+=blockDim.x * gridDim.x) {
        size_t cell_index = i * data_dims.cell_dim;
        unsigned short new_pop = calc_cell_population(cell_index, data_dims, ghost_cols, data);
        result_data[i+7] = new_pop;
    }
}


void launch_kernel(int rank, int thread_count, struct DataDims data_dims, struct GhostCols ghost_cols,
        unsigned short *data, unsigned short *result_data) {

    // Get the number of cuda devices and set the current device to the rank modulo the number of devices
    hipError_t cuda_error;
    int cuda_device_count;
    if( (cuda_error = hipGetDeviceCount( &cuda_device_count)) != hipSuccess ) {
        printf(" Unable to determine cuda device count, error is %d, count is %d\n", cuda_error, cuda_device_count );
        exit(cuda_error);
    }
    if( (cuda_error = hipSetDevice( rank % cuda_device_count )) != hipSuccess ) {
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n", rank, (rank % cuda_device_count), cuda_error);
        exit(cuda_error);
    }

    // Determine how many blocks should be allocated to the kernel with a maximum of 65535
    size_t block_count = (data_dims.row_dim * data_dims.col_dim) / thread_count + 1;
    block_count = block_count > 65535 ? 65535 : block_count;
    // Launch the kernel with the determined block count and thread count
    cuda_kernel<<<block_count, (size_t) thread_count>>>(data_dims, ghost_cols, data, result_data);
    // Synchronize the device after each launch
    hipDeviceSynchronize();
}
